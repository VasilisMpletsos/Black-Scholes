#include "hip/hip_runtime.h"
#define DATA_SIZE 858

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>  
#include "BlackScholes_kernel.cuh"

const int REPEAT_ITERATIONS_EXPERIMENT = 1000;


const int MEMORY_SIZE_ALLOCATION_FLOAT = DATA_SIZE * sizeof(float);
const int MEMORY_SIZE_ALLOCATION_INT = DATA_SIZE * sizeof(float);
const float RISKFREE = 0.01575f;
const float VOLATILITY = 0.25f;

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

////////////////////////////////////////////////////////////////////////////////
// ------------------------------ Main program ------------------------------ //
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("Starting BlackScholes on GPU...\n");

    float *h_OptionResultGPULarge, *h_StockPrice, *h_OptionStrike, *h_OptionYears;
    float *d_OptionResult, *d_StockPrice, *d_OptionStrike, *d_OptionYears;
    int *h_OptionTypes, *d_OptionTypes;

    double gpuTime;

    StopWatchInterface *hTimer = NULL;
    int i;

    // Detect NVIDIA GPU
    int device = findCudaDevice(argc, (const char **)argv);

    sdkCreateTimer(&hTimer);

    // Get gpu properties
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));
    printf("GPU max threads per block %d \n", deviceProp.maxThreadsPerBlock);

    int MAX_THREADS_PER_BLOCK = deviceProp.maxThreadsPerBlock;
    int numSMs = deviceProp.multiProcessorCount;
    int maxThreadsPerSM = deviceProp.maxThreadsPerMultiProcessor;
    int blockSize = deviceProp.maxThreadsPerBlock;
    int blocksPerSM = DIV_UP(maxThreadsPerSM, blockSize);
    printf("GPU SM count: %d\n", numSMs);
    printf("Max threads per SM: %d\n", maxThreadsPerSM);
    printf("Using block size: %d\n", blockSize);
    printf("Blocks per SM for occupancy: %d\n", blocksPerSM);

    // Set oversubscription factor to make kernel run longer for accurate timing
    const int overSub = 32;
    int minNumBlocks = numSMs * blocksPerSM * overSub;
    size_t minTotalOptions = (size_t)minNumBlocks * blockSize;
    size_t numReplicas = DIV_UP(minTotalOptions, DATA_SIZE);
    size_t totalOptions = numReplicas * DATA_SIZE;
    int numBlocks = DIV_UP(totalOptions, blockSize);
    printf("Using %zu replicas, total options: %zu\n", numReplicas, totalOptions);
    printf("Launching %d blocks of %d threads\n", numBlocks, blockSize);

    printf("Allocating CPU memory for options.\n");
    h_StockPrice = (float *)malloc(MEMORY_SIZE_ALLOCATION_FLOAT);
    h_OptionStrike = (float *)malloc(MEMORY_SIZE_ALLOCATION_FLOAT);
    h_OptionYears = (float *)malloc(MEMORY_SIZE_ALLOCATION_FLOAT);
    h_OptionTypes = (int *)malloc(MEMORY_SIZE_ALLOCATION_INT);

    printf("Reading data...\n");
    // Reading data from files
    std::cout << "Reading data...\n";
    std::ifstream closeFile("./datasets/option_price.txt");
    std::ifstream strikeFile("./datasets/strike.txt");
    std::ifstream tteFile("./datasets/tte.txt");
    std::ifstream typeFile("./datasets/type.txt");

    // Check if files opened successfully
    if (!closeFile || !strikeFile || !tteFile || !typeFile) {
        throw std::runtime_error("Failed to open one or more input files.");
    }

    // Load data into host arrays
    for (int i = 0; i < DATA_SIZE; i++) {
        std::string line;

        std::getline(closeFile, line);
        h_StockPrice[i] = std::stof(line);

        std::getline(strikeFile, line);
        h_OptionStrike[i] = std::stof(line);

        std::getline(tteFile, line);
        h_OptionYears[i] = std::stof(line);

        std::getline(typeFile, line);
        h_OptionTypes[i] = std::stoi(line);  // Assuming type is an integer
    }

    // Close files
    closeFile.close();
    strikeFile.close();
    tteFile.close();
    typeFile.close();

    // Allocate large host arrays for replication
    size_t memSizeFloat = totalOptions * sizeof(float);
    size_t memSizeInt = totalOptions * sizeof(int);
    float *h_StockPriceLarge = (float *)malloc(memSizeFloat);
    float *h_OptionStrikeLarge = (float *)malloc(memSizeFloat);
    float *h_OptionYearsLarge = (float *)malloc(memSizeFloat);
    int *h_OptionTypesLarge = (int *)malloc(memSizeInt);
    h_OptionResultGPULarge = (float *)malloc(memSizeFloat);

    // Replicate the data
    for (size_t r = 0; r < numReplicas; r++) {
        memcpy(h_StockPriceLarge + r * DATA_SIZE, h_StockPrice, MEMORY_SIZE_ALLOCATION_FLOAT);
        memcpy(h_OptionStrikeLarge + r * DATA_SIZE, h_OptionStrike, MEMORY_SIZE_ALLOCATION_FLOAT);
        memcpy(h_OptionYearsLarge + r * DATA_SIZE, h_OptionYears, MEMORY_SIZE_ALLOCATION_FLOAT);
        memcpy(h_OptionTypesLarge + r * DATA_SIZE, h_OptionTypes, MEMORY_SIZE_ALLOCATION_INT);
    }

    // Free original small host input arrays to save memory
    free(h_StockPrice);
    free(h_OptionStrike);
    free(h_OptionYears);
    free(h_OptionTypes);

    printf("Allocating GPU memory for options.\n");
    checkCudaErrors(hipMalloc((void **)&d_OptionResult, memSizeFloat));
    checkCudaErrors(hipMalloc((void **)&d_StockPrice, memSizeFloat));
    checkCudaErrors(hipMalloc((void **)&d_OptionStrike, memSizeFloat));
    checkCudaErrors(hipMalloc((void **)&d_OptionYears, memSizeFloat));
    checkCudaErrors(hipMalloc((void **)&d_OptionTypes, memSizeInt));

    printf("Copying input data from host CPU to GPU registers.\n");
    checkCudaErrors(hipMemcpy(d_StockPrice,  h_StockPriceLarge,  memSizeFloat, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionStrike, h_OptionStrikeLarge, memSizeFloat, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionYears,  h_OptionYearsLarge,  memSizeFloat, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionTypes,  h_OptionTypesLarge,  memSizeInt, hipMemcpyHostToDevice));
    printf("Data copies successfully.\n\n");

    // Initialize result
    for (i = 0; i < totalOptions; i++)
    {
        h_OptionResultGPULarge[i] = 0.0f;
    }

    printf("Executing Black-Scholes GPU kernel %i iterations...\n", REPEAT_ITERATIONS_EXPERIMENT);
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);

    for (i = 0; i < REPEAT_ITERATIONS_EXPERIMENT; i++)
    {
        BlackScholesGPU<<<numBlocks, blockSize>>>(
            (int1 *)d_OptionTypes,
            (float1 *)d_StockPrice,
            (float1 *)d_OptionStrike,
            RISKFREE,
            VOLATILITY,
            (float1 *)d_OptionYears,
            (float1 *)d_OptionResult
        );
        getLastCudaError("BlackScholesGPU() execution failed\n");
    }

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    gpuTime = sdkGetTimerValue(&hTimer) / REPEAT_ITERATIONS_EXPERIMENT;

    //Both call and put is calculated
    printf("Black Scholes GPU() average execution time: %f msec\n", gpuTime * 1000);
    printf("Effective memory bandwidth: %f GB/s\n", ((double)(5 * totalOptions * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
    printf("Gigaoptions per second: %f \n\n", ((double)(totalOptions) * 1E-9) / (gpuTime * 1E-3));

    printf("\nReading back GPU results...\n");
    //Read back GPU results to compare them to CPU results
    checkCudaErrors(hipMemcpy(h_OptionResultGPULarge, d_OptionResult, memSizeFloat, hipMemcpyDeviceToHost));

    // Iterate through first set of results and print
    // for (int i = 0; i < DATA_SIZE; i++) {
    //     printf("Option %d: %.5f\n", i+1, h_OptionResultGPULarge[i]);
    // }

    printf("Cleaning GPU allocated memory.\n");
    checkCudaErrors(hipFree(d_OptionYears));
    checkCudaErrors(hipFree(d_OptionStrike));
    checkCudaErrors(hipFree(d_StockPrice));
    checkCudaErrors(hipFree(d_OptionResult));
    checkCudaErrors(hipFree(d_OptionTypes));

    printf("Cleaning CPU allocated memory\n");
    free(h_OptionYearsLarge);
    free(h_OptionStrikeLarge);
    free(h_StockPriceLarge);
    free(h_OptionTypesLarge);
    free(h_OptionResultGPULarge);
    sdkDeleteTimer(&hTimer);
    printf("Test Done\n");
    exit(EXIT_SUCCESS);
}