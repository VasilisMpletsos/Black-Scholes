#include "hip/hip_runtime.h"
#define DATA_SIZE 858

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>  
#include "BlackScholes_kernel.cuh"

const int REPEAT_ITERATIONS_EXPERIMENT = 1000;


const int MEMORY_SIZE_ALLOCATION_FLOAT = DATA_SIZE * sizeof(float);
const int MEMORY_SIZE_ALLOCATION_INT = DATA_SIZE * sizeof(float);
const float RISKFREE = 0.01575f;
const float VOLATILITY = 0.25f;

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

////////////////////////////////////////////////////////////////////////////////
// ------------------------------ Main program ------------------------------ //
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("Starting BlackScholes on GPU...\n");

    float *h_OptionResultGPU, *h_StockPrice, *h_OptionStrike, *h_OptionYears;
    float *d_OptionResult, *d_StockPrice, *d_OptionStrike, *d_OptionYears;
    int *h_OptionTypes, *d_OptionTypes;

    double gpuTime;

    StopWatchInterface *hTimer = NULL;
    int i;

    // Detect NVIDIA GPU
    findCudaDevice(argc, (const char **)argv);

    sdkCreateTimer(&hTimer);

    printf("Allocating CPU memory for options.\n");
    h_OptionResultGPU  = (float *)malloc(MEMORY_SIZE_ALLOCATION_FLOAT);
    h_StockPrice = (float *)malloc(MEMORY_SIZE_ALLOCATION_FLOAT);
    h_OptionStrike = (float *)malloc(MEMORY_SIZE_ALLOCATION_FLOAT);
    h_OptionYears = (float *)malloc(MEMORY_SIZE_ALLOCATION_FLOAT);
    h_OptionTypes = (int *)malloc(MEMORY_SIZE_ALLOCATION_INT);

    printf("Allocating GPU memory for options.\n");
    checkCudaErrors(hipMalloc((void **)&d_OptionResult, MEMORY_SIZE_ALLOCATION_FLOAT));
    checkCudaErrors(hipMalloc((void **)&d_StockPrice, MEMORY_SIZE_ALLOCATION_FLOAT));
    checkCudaErrors(hipMalloc((void **)&d_OptionStrike, MEMORY_SIZE_ALLOCATION_FLOAT));
    checkCudaErrors(hipMalloc((void **)&d_OptionYears, MEMORY_SIZE_ALLOCATION_FLOAT));
    checkCudaErrors(hipMalloc((void **)&d_OptionTypes, MEMORY_SIZE_ALLOCATION_INT));

    printf("Reading data...\n");
    // Reading data from files
    std::cout << "Reading data...\n";
    std::ifstream closeFile("./datasets/option_price.txt");
    std::ifstream strikeFile("./datasets/strike.txt");
    std::ifstream tteFile("./datasets/tte.txt");
    std::ifstream typeFile("./datasets/type.txt");

    // Check if files opened successfully
    if (!closeFile || !strikeFile || !tteFile || !typeFile) {
        throw std::runtime_error("Failed to open one or more input files.");
    }

    // Load data into host arrays
    for (int i = 0; i < DATA_SIZE; i++) {
        std::string line;

        std::getline(closeFile, line);
        h_StockPrice[i] = std::stof(line);

        std::getline(strikeFile, line);
        h_OptionStrike[i] = std::stof(line);

        std::getline(tteFile, line);
        h_OptionYears[i] = std::stof(line);

        std::getline(typeFile, line);
        h_OptionTypes[i] = std::stoi(line);  // Assuming type is an integer
    }

    // Close files
    closeFile.close();
    strikeFile.close();
    tteFile.close();
    typeFile.close();

    //Generate random options set
    for (i = 0; i < DATA_SIZE; i++)
    {
        h_OptionResultGPU[i] = 0.0f;
    }

    printf("Copying input data from host CPU to GPU registers.\n");
    checkCudaErrors(hipMemcpy(d_StockPrice,  h_StockPrice,   MEMORY_SIZE_ALLOCATION_FLOAT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionStrike, h_OptionStrike,  MEMORY_SIZE_ALLOCATION_FLOAT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionYears,  h_OptionYears,   MEMORY_SIZE_ALLOCATION_FLOAT, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionTypes,  h_OptionTypes,   MEMORY_SIZE_ALLOCATION_INT, hipMemcpyHostToDevice));
    printf("Data copies successfully.\n\n");


    printf("Executing Black-Scholes GPU kernel %i iterations...\n", REPEAT_ITERATIONS_EXPERIMENT);
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);

    for (i = 0; i < REPEAT_ITERATIONS_EXPERIMENT; i++)
    {
        // because we have 858 options, we need to launch 13 blocks of 66 threads
        BlackScholesGPU<<<13, 66>>>(
            (int1 *)d_OptionTypes,
            (float1 *)d_StockPrice,
            (float1 *)d_OptionStrike,
            RISKFREE,
            VOLATILITY,
            (float1 *)d_OptionYears,
            (float1 *)d_OptionResult
        );
        getLastCudaError("BlackScholesGPU() execution failed\n");
    }

    sdkStopTimer(&hTimer);
    checkCudaErrors(hipDeviceSynchronize());
    gpuTime = sdkGetTimerValue(&hTimer) / REPEAT_ITERATIONS_EXPERIMENT;

    //Both call and put is calculated
    printf("Black Scholes GPU() average execution time: %f msec\n", gpuTime);
    printf("Effective memory bandwidth: %f GB/s\n", ((double)(5 * DATA_SIZE * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
    printf("Gigaoptions per second: %f \n\n", ((double)(DATA_SIZE) * 1E-9) / (gpuTime * 1E-3));

    printf("\nReading back GPU results...\n");
    //Read back GPU results to compare them to CPU results
    checkCudaErrors(hipMemcpy(h_OptionResultGPU, d_OptionResult, MEMORY_SIZE_ALLOCATION_FLOAT, hipMemcpyDeviceToHost));

    // // Iterate through results and print
    // for (int i = 0; i < DATA_SIZE; i++) {
    //     printf("Option %d: %.5f\n", i+1, h_OptionResultGPU[i]);
    // }

    printf("Cleaning GPU allocated memory.\n");
    checkCudaErrors(hipFree(d_OptionYears));
    checkCudaErrors(hipFree(d_OptionStrike));
    checkCudaErrors(hipFree(d_StockPrice));
    checkCudaErrors(hipFree(d_OptionResult));
    checkCudaErrors(hipFree(d_OptionTypes));

    printf("Cleaning CPU allocated memory\n");
    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_OptionTypes);
    free(h_OptionResultGPU);
    sdkDeleteTimer(&hTimer);
    printf("Test Done\n");
    exit(EXIT_SUCCESS);
}
