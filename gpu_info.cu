#include <stdio.h>
#define KB 1024
#define MB 1024 * KB
#define GB 1024 * MB

int main() {
    // Read cuda device properties
    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    // Extract device properties
    size_t totalGlobalMem = props.totalGlobalMem;
    size_t sharedMemPerBlock = props.sharedMemPerBlock;
    int threads_per_block = props.maxThreadsPerBlock;
    int threads_per_multiprocessor = props.maxThreadsPerMultiProcessor;
    int registers_per_block = props.regsPerBlock;
    int maxThreadsDim[3] = {props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2]};
    int maxGridSize[3] = {props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]};
    int clockRate = props.clockRate;

    // Print device properties
    printf("Maximum threads per block: %d\n", threads_per_block);
    printf("Maximum threads per multiprocessor: %d\n", threads_per_multiprocessor);
    printf("Registers per block: %d\n", registers_per_block);
    printf("Shared memory per block: %zu MB\n", sharedMemPerBlock/MB);
    printf("Total global memory: %zu GB\n", totalGlobalMem/(1024*1024*1024));
    printf("Maximum dimension of each block:");
    printf(" x: %d, y: %d, z: %d\n", maxThreadsDim[0], maxThreadsDim[1], maxThreadsDim[2]);
    printf("Maximum dimension of each grid:");
    printf(" x: %d, y: %d, z: %d\n", maxGridSize[0], maxGridSize[1], maxGridSize[2]);
    printf("Clock rate: %d MHz\n", clockRate/1000);

    return 0;
}